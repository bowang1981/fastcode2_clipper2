#include "hip/hip_runtime.h"
#include "clipper2/clipper.offset.cuh"
#include "clipper2/clipper.core.cuh"


namespace Clipper2Lib {

__device__ void Append(cuPath64& input, int64_t x, int64_t y)
{
	input.points[input.size].x = x;
	input.points[input.size].y = y;
	input.size = input.size + 1;
}
/*
 *  We need port the following functions into kernel functions
__device__ void ClipperOffset::DoBevel(cuPath64* input, size_t j, size_t k, cuPath64* output)
{
	PointD pt1, pt2;
	if (j == k)
	{
		double abs_delta = std::abs(group_delta_);
		pt1 = PointD(path[j].x - abs_delta * norms[j].x, path[j].y - abs_delta * norms[j].y);
		pt2 = PointD(path[j].x + abs_delta * norms[j].x, path[j].y + abs_delta * norms[j].y);
	}
	else
	{
		pt1 = PointD(path[j].x + group_delta_ * norms[k].x, path[j].y + group_delta_ * norms[k].y);
		pt2 = PointD(path[j].x + group_delta_ * norms[j].x, path[j].y + group_delta_ * norms[j].y);
	}
	path_out.push_back(Point64(pt1));
	path_out.push_back(Point64(pt2));
}

void ClipperOffset::DoSquare(const Path64& path, size_t j, size_t k)
{
	PointD vec;
	if (j == k)
		vec = PointD(norms[j].y, -norms[j].x);
	else
		vec = GetAvgUnitVector(
			PointD(-norms[k].y, norms[k].x),
			PointD(norms[j].y, -norms[j].x));

	double abs_delta = std::abs(group_delta_);

	// now offset the original vertex delta units along unit vector
	PointD ptQ = PointD(path[j]);
	ptQ = TranslatePoint(ptQ, abs_delta * vec.x, abs_delta * vec.y);
	// get perpendicular vertices
	PointD pt1 = TranslatePoint(ptQ, group_delta_ * vec.y, group_delta_ * -vec.x);
	PointD pt2 = TranslatePoint(ptQ, group_delta_ * -vec.y, group_delta_ * vec.x);
	// get 2 vertices along one edge offset
	PointD pt3 = GetPerpendicD(path[k], norms[k], group_delta_);
	if (j == k)
	{
		PointD pt4 = PointD(pt3.x + vec.x * group_delta_, pt3.y + vec.y * group_delta_);
		PointD pt = IntersectPoint(pt1, pt2, pt3, pt4);

		//get the second intersect point through reflecion
		path_out.push_back(Point64(ReflectPoint(pt, ptQ)));
		path_out.push_back(Point64(pt));
	}
	else
	{
		PointD pt4 = GetPerpendicD(path[j], norms[k], group_delta_);
		PointD pt = IntersectPoint(pt1, pt2, pt3, pt4);

		path_out.push_back(Point64(pt));
		//get the second intersect point through reflecion
		path_out.push_back(Point64(ReflectPoint(pt, ptQ)));
	}
}

void ClipperOffset::DoMiter(const Path64& path, size_t j, size_t k, double cos_a)
{
	double q = group_delta_ / (cos_a + 1);

	path_out.push_back(Point64(
		path[j].x + (norms[k].x + norms[j].x) * q,
		path[j].y + (norms[k].y + norms[j].y) * q));
}

void ClipperOffset::DoRound(const Path64& path, size_t j, size_t k, double angle)
{
	if (deltaCallback64_) {
		// when deltaCallback64_ is assigned, group_delta_ won't be constant,
		// so we'll need to do the following calculations for *every* vertex.
		double abs_delta = std::fabs(group_delta_);
		double arcTol = (arc_tolerance_ > floating_point_tolerance ?
			std::min(abs_delta, arc_tolerance_) :
			std::log10(2 + abs_delta) * default_arc_tolerance);
		double steps_per_360 = std::min(PI / std::acos(1 - arcTol / abs_delta), abs_delta * PI);
		step_sin_ = std::sin(2 * PI / steps_per_360);
		step_cos_ = std::cos(2 * PI / steps_per_360);
		if (group_delta_ < 0.0) step_sin_ = -step_sin_;
		steps_per_rad_ = steps_per_360 / (2 * PI);
	}

	Point64 pt = path[j];
	PointD offsetVec = PointD(norms[k].x * group_delta_, norms[k].y * group_delta_);

	if (j == k) offsetVec.Negate();

	path_out.push_back(Point64(pt.x + offsetVec.x, pt.y + offsetVec.y));
	int steps = static_cast<int>(std::ceil(steps_per_rad_ * std::abs(angle))); // #448, #456
	for (int i = 1; i < steps; ++i) // ie 1 less than steps
	{
		offsetVec = PointD(offsetVec.x * step_cos_ - step_sin_ * offsetVec.y,
			offsetVec.x * step_sin_ + offsetVec.y * step_cos_);

		path_out.push_back(Point64(pt.x + offsetVec.x, pt.y + offsetVec.y));
	}
	path_out.push_back(GetPerpendic(path[j], norms[j], group_delta_));
}

void ClipperOffset::OffsetPoint(Group& group, const Path64& path, size_t j, size_t k)
{
	// Let A = change in angle where edges join
	// A == 0: ie no change in angle (flat join)
	// A == PI: edges 'spike'
	// sin(A) < 0: right turning
	// cos(A) < 0: change in angle is more than 90 degree

	if (path[j] == path[k]) { k = j; return; }

	double sin_a = CrossProduct(norms[j], norms[k]);
	double cos_a = DotProduct(norms[j], norms[k]);
	if (sin_a > 1.0) sin_a = 1.0;
	else if (sin_a < -1.0) sin_a = -1.0;

	if (deltaCallback64_) {
		group_delta_ = deltaCallback64_(path, norms, j, k);
		if (group.is_reversed) group_delta_ = -group_delta_;
	}
	if (std::fabs(group_delta_) <= floating_point_tolerance)
	{
		path_out.push_back(path[j]);
		return;
	}

	if (cos_a > -0.99 && (sin_a * group_delta_ < 0)) // test for concavity first (#593)
	{
		// is concave
		path_out.push_back(GetPerpendic(path[j], norms[k], group_delta_));
		// this extra point is the only (simple) way to ensure that
	  // path reversals are fully cleaned with the trailing clipper
		path_out.push_back(path[j]); // (#405)
		path_out.push_back(GetPerpendic(path[j], norms[j], group_delta_));
	}
	else if (cos_a > 0.999 && join_type_ != JoinType::Round)
	{
		// almost straight - less than 2.5 degree (#424, #482, #526 & #724)
		DoMiter(path, j, k, cos_a);
	}
	else if (join_type_ == JoinType::Miter)
	{
		// miter unless the angle is sufficiently acute to exceed ML
		if (cos_a > temp_lim_ - 1) DoMiter(path, j, k, cos_a);
		else DoSquare(path, j, k);
	}
	else if (join_type_ == JoinType::Round)
		DoRound(path, j, k, std::atan2(sin_a, cos_a));
	else if ( join_type_ == JoinType::Bevel)
		DoBevel(path, j, k);
	else
		DoSquare(path, j, k);
}

void ClipperOffset::OffsetPolygon(Group& group, const Path64& path)
{
	path_out.clear();
	for (Path64::size_type j = 0, k = path.size() -1; j < path.size(); k = j, ++j)
		OffsetPoint(group, path, j, k);
	solution.push_back(path_out);
}
*/

void offset_execute(const Paths64& input, const Rect64& rect, Paths64& output)
{

}

} // end of namespace
